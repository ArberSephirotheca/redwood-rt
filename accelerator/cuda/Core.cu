#include "hip/hip_runtime.h"
#include <>

#include <limits>

#include "CudaUtils.cuh"
#include "hip/hip_runtime.h"

hipStream_t streams[kNumStreams];
bool stream_created = false;

__global__ void CudaWarmup() {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

namespace redwood {
namespace accelerator {

void Initialization() {
  CudaWarmup<<<1, 1024>>>();
  HANDLE_ERROR(hipDeviceSynchronize());
}

// CUDA Only
void AttachStreamMem(const int stream_id, void* addr) {
  if (!stream_created) {
    for (unsigned i = 0; i < kNumStreams; i++) {
      HANDLE_ERROR(hipStreamCreate(&streams[i]));
    }
    stream_created = true;
  }

  hipStreamAttachMemAsync(streams[stream_id], addr);
}

void DeviceSynchronize() { HANDLE_ERROR(hipDeviceSynchronize()); }

void DeviceStreamSynchronize(const int stream_id) {
  HANDLE_ERROR(hipStreamSynchronize(streams[stream_id]));
}
}
}  // namespace redwood::accelerator