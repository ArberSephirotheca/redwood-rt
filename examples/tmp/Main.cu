#include "hip/hip_runtime.h"

#include <hip/hip_cooperative_groups.h>
#include <>

#include <algorithm>
#include <chrono>
#include <cmath>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include "../Utils.hpp"
#include "CuUtils.hpp"
#include "hip/hip_runtime.h"

namespace cg = cooperative_groups;

constexpr auto kNumBlocks = 1;

// template <typename T>
// void check(T result, char const* const func, const char* const file,
//            int const line) {
//   if (result) {
//     fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
//             static_cast<unsigned int>(result), _cudaGetErrorEnum(result),
//             func);
//     exit(EXIT_FAILURE);
//   }
// }

// #define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

struct EuclideanFunctor {
  __host__ __device__ __forceinline__ float operator()(const float2& p,
                                                       const float2& q) const {
    const auto dx = p.x - q.x;
    const auto dy = p.y - q.y;
    return sqrtf(dx * dx + dy * dy);
  }
};

// too many resources requested for launch
struct HaversineFunctor {
  __host__ __device__ __forceinline__ float operator()(const float2& p,
                                                       const float2& q) const {
    auto lat1 = p.x;
    auto lat2 = q.x;
    const auto lon1 = p.y;
    const auto lon2 = q.y;

    const auto dLat = (lat2 - lat1) * M_PI / 180.0f;
    const auto dLon = (lon2 - lon1) * M_PI / 180.0f;

    // convert to radians
    lat1 = lat1 * M_PI / 180.0f;
    lat2 = lat2 * M_PI / 180.0f;

    // apply formula
    float a = powf(sinf(dLat / 2), 2) +
              powf(sinf(dLon / 2), 2) * cosf(lat1) * cosf(lat2);
    constexpr float rad = 6371;
    float c = 2 * asinf(sqrtf(a));
    return rad * c;
  }
};

template <typename DataT, typename ResultT, typename ReductionOp>
__device__ void FunctionKernel(cg::thread_group g, DataT* u_buffer,
                               ResultT* u_result, const DataT q) {
  // This need to be a conexpr, because I am passing this as a template argument
  // for Cub library. Although is is just the same as 'g.size()'
  constexpr int block_threads = 1024;
  constexpr int items_to_reduce = 1024;
  constexpr int items_per_thread = items_to_reduce / block_threads;

  using BlockLoad = hipcub::BlockLoad<DataT, block_threads, items_per_thread,
                                   hipcub::BLOCK_LOAD_STRIPED>;
  using BlockReduce = hipcub::BlockReduce<ResultT, block_threads>;

  constexpr auto functor = ReductionOp();

  __shared__ union {
    typename BlockLoad::TempStorage load;
    typename BlockReduce::TempStorage reduce;
  } temp_storage;

  const auto tid = g.thread_rank();

  DataT thread_data[items_per_thread];
  ResultT thread_value[items_per_thread];

  BlockLoad(temp_storage.load).Load(u_buffer, thread_data);

#pragma unroll
  for (int i = 0; i < items_per_thread; ++i) {
    thread_value[i] = functor(thread_data[i], q);
  }

  ResultT aggregate =
      BlockReduce(temp_storage.reduce).Reduce(thread_value, hipcub::Min());

  // Final step reduction
  if (tid == 0) u_result[0] = min(u_result[0], aggregate);
}

__device__ __forceinline__ void WaitCPU(int* com) {
  int block_id = blockIdx.x;
  while (com[block_id] != 1 && com[kNumBlocks] != 1) {
    // __threadfence_system(); ?
    __threadfence();
  }
}

__device__ __forceinline__ void WorkComplete(int* com) {
  int block_id = blockIdx.x;
  com[block_id] = 0;
}

template <typename DataT, typename ResultT,
          typename ReductionOp = EuclideanFunctor>
__global__ void PersistentKernel(DataT* u_buffer, const DataT q,
                                 ResultT* u_result, int* com) {
  auto cta = cg::this_thread_block();
  const auto tid = cta.thread_rank();

  while (com[kNumBlocks] != 1) {
    if (tid == 0) WaitCPU(com);
    __syncthreads();

    // cancelling point
    if (com[kNumBlocks] == 1) return;

    FunctionKernel<DataT, ResultT, ReductionOp>(cta, u_buffer, u_result, q);

    if (tid == 0) WorkComplete(com);
  }
}

// template <typename DataT, typename ResultT>
// __global__ void PersistentKernelDoubleBuffer(DataT* u_buffer_a,    //
//                                              DataT* u_buffer_b,    //
//                                              const int n,          //
//                                              const DataT q,        //
//                                              ResultT* u_result_a,  //
//                                              ResultT* u_result_b,  //
//                                              int* com) {
//   auto cta = cg::this_thread_block();
//   const auto tid = cta.thread_rank();

//   // Temporary.
//   // com[0] = buffer a status
//   //    0 : buffer a
//   //    1 : buffer b
//   // com[2] = program status

//   while (com[2] != 1) {
//     if (tid == 0) WaitCPU(com);
//     __syncthreads();

//     // cancelling point
//     if (com[2] == 1) return;

//     FunctionKernel(cta, u_buffer, n, u_result, q);

//     if (tid == 0) WorkComplete(com);
//   }
// }

template <typename DataT, typename ResultT,
          typename ReductionOp = EuclideanFunctor>
__global__ void NormalKernel(DataT* d_data, const int n, const DataT q,
                             ResultT* u_result) {
  auto cta = cg::this_thread_block();
  const auto tid = cta.thread_rank();
  const int block_threads = cta.size();

  const auto iterations = n / block_threads;
  for (int i = 0; i < iterations; ++i) {
    if (tid == 0) printf("gpu iteration (%d/%d)\n", i, iterations);
    FunctionKernel<DataT, ResultT, ReductionOp>(cta, d_data + i * block_threads,
                                                u_result, q);
  }
}

void StartGPU(int* com) {
  // atomic?
  for (int i = 0; i < kNumBlocks; ++i) com[i] = 1;
}

void WaitGPU(int* com) {
  int sum;
  do {
    sum = 0;
    asm volatile("" ::: "memory");
    for (int i = 0; i < kNumBlocks; ++i) sum |= com[i];
  } while (sum != 0);
}

void EndGPU(int* com) {
  printf("cpu is ending GPU\n");
  com[kNumBlocks] = 1;
}

static float2 RandomPoint() {
  static thread_local std::mt19937 generator(114514);  // NOLINT(cert-msc51-cpp)
  static std::uniform_real_distribution<float> lat_dist(-90.0f, 90.0f);
  static std::uniform_real_distribution<float> lon_dist(-180.0f, 180.0f);

  return {lat_dist(generator), lon_dist(generator)};
}

std::vector<float2> GenerateRandomPoints(const int num_points) {
  std::vector<float2> points(num_points);
  std::generate(points.begin(), points.end(), RandomPoint);
  return points;
}

float2* tmp = nullptr;

int main(int argc, char** argv) {
  if (argc < 2) {
    std::cerr << "Bad argument\n";
    return EXIT_FAILURE;
  }

  const int enable_pk = atoi(argv[1]);

  constexpr int n = 1024 * 1024;
  constexpr int buffer_size = 1024;
  constexpr auto num_threads = 1024;

  float2* u_buffer = nullptr;
  float* u_result = nullptr;
  int* u_com = nullptr;

  cudaAllocMapped(&u_buffer, sizeof(float2) * buffer_size);
  cudaAllocMapped(&u_result, sizeof(float) * 1);
  cudaAllocMapped(&u_com, sizeof(int) * (kNumBlocks + 1));

  u_result[0] = std::numeric_limits<float>::max();

  auto h_p_data = GenerateRandomPoints(n);
  const float2 q{0.0f, 0.0f};

  TimeTask("CPU Compute: ", [&] {
    auto sum = std::numeric_limits<float>::max();

    constexpr auto functor = HaversineFunctor();
    for (int i = 0; i < n; ++i) {
      const auto dist = functor(h_p_data[i], q);
      sum = std::min(sum, dist);
    }

    std::cout << "Ground truth: " << sum << std::endl;
  });

  tmp = (float2*)malloc(sizeof(float2) * n);
  TimeTask("CPU Memcpy: ",
           [&] { memcpy(tmp, h_p_data.data(), sizeof(float2) * n); });

  if (enable_pk) {
    // Launching the PK only once
    PersistentKernel<<<kNumBlocks, num_threads>>>(u_buffer, q, u_result, u_com);

    TimeTask("PK GPU: ", [&] {
      const auto iterations = n / num_threads;
      for (int i = 0; i < iterations; ++i) {
        // std::cout << "\nIteration: (" << i << '/' << iterations << ')' <<
        // std::endl;
        memcpy(u_buffer, h_p_data.data() + i * num_threads,
               sizeof(float2) * num_threads);

        StartGPU(u_com);

        // CPU Works here...

        WaitGPU(u_com);
      }
    });

    EndGPU(u_com);

    HANDLE_ERROR(hipHostFree(u_buffer));
    HANDLE_ERROR(hipHostFree(u_com));

  } else {
    float2* d_data = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&d_data, sizeof(float2) * n));
    HANDLE_ERROR(hipDeviceSynchronize());

    TimeTask("Normal GPU (1 block) memcpy: ", [&] {
      HANDLE_ERROR(hipMemcpy(d_data, h_p_data.data(), sizeof(float2) * n,
                              hipMemcpyHostToDevice));
    });

    TimeTask("Normal GPU (1 block) compute: ", [&] {
      NormalKernel<float2, float, HaversineFunctor>
          <<<kNumBlocks, num_threads>>>(d_data, n, q, u_result);
      HANDLE_ERROR(hipGetLastError());

      HANDLE_ERROR(hipDeviceSynchronize());
    });
  }

  std::cout << "\tu_result: " << u_result[0] << std::endl;

  HANDLE_ERROR(hipHostFree(u_result));

  return 0;
}