#include <iostream>

#include "Redwood.hpp"

//
#include "CudaUtils.cuh"
#include "Kernels.hpp"
#include "UserKernels.cuh"

namespace redwood {

constexpr auto kNumStreams = 2;

hipStream_t streams[kNumStreams];
bool stream_created = false;

// --------- Core ---------
void Init() {
  CudaWarmup<<<1, 1024>>>();
  DeviceSynchronize();
}

void DeviceSynchronize() { HANDLE_ERROR(hipDeviceSynchronize()); }

void DeviceStreamSynchronize(const int stream_id) {
  HANDLE_ERROR(hipStreamSynchronize(streams[stream_id]));
}

void AttachStreamMem(const int stream_id, void* addr) {
  if (!stream_created) {
    for (int i = 0; i < kNumStreams; i++) {
      HANDLE_ERROR(hipStreamCreate(&streams[i]));
    }
    stream_created = true;
  }

  hipStreamAttachMemAsync(streams[stream_id], addr);
}

// --------- Unified Memory ---------

void* UsmMalloc(std::size_t n) {
  void* tmp;
  HANDLE_ERROR(hipMallocManaged(&tmp, n));
  std::cout << "accelerator::UsmMalloc() " << tmp << ": " << n << " bytes."
            << std::endl;
  return tmp;
}

void UsmFree(void* ptr) {
  std::cout << "accelerator::UsmFree() " << ptr << std::endl;
  if (ptr) {
    HANDLE_ERROR(hipFree(ptr));
  }
}

// --------- Kernel related ---------

void LaunchNnKenrnel(const int* u_leaf_indices,  /**/
                     const Point4F* u_q_points,  /**/
                     const int num_active_leafs, /**/
                     float* u_out,               /* stream base addr */
                     const Point4F* u_lnt_data,  /**/
                     const int max_leaf_size, const int stream_id) {
  // FindMinDistWarp6<<<1, 1024, 0, streams[stream_id]>>>(
  //     u_leaf_indices, u_q_points, num_active_leafs, u_out, u_lnt_data,
  //     max_leaf_size);

  CudaNaive<<<1, 1024, 0, streams[stream_id]>>>(u_leaf_indices, u_q_points,
                                                num_active_leafs, u_out,
                                                u_lnt_data, max_leaf_size);

  // CudaNnDebug<<<1, 1, 0, streams[stream_id]>>>(u_leaf_indices, u_q_points,
  //  num_active_leafs, u_out,
  //  u_lnt_data, max_leaf_size);
}

}  // namespace redwood