#include "hip/hip_runtime.h"

#include <hip/hip_cooperative_groups.h>
#include <>

#include <algorithm>
#include <chrono>
#include <cmath>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include "../Utils.hpp"
#include "CuUtils.hpp"
#include "hip/hip_runtime.h"

namespace cg = cooperative_groups;

constexpr auto kNumBlocks = 1;

namespace Distance {

struct EuclideanFunctor {
  __host__ __device__ __forceinline__ float operator()(const float2& p,
                                                       const float2& q) const {
    const auto dx = p.x - q.x;
    const auto dy = p.y - q.y;
    return sqrtf(dx * dx + dy * dy);
  }
};

// too many resources requested for launch
struct HaversineFunctor {
  __host__ __device__ __forceinline__ float operator()(const float2& p,
                                                       const float2& q) const {
    auto lat1 = p.x;
    auto lat2 = q.x;
    const auto lon1 = p.y;
    const auto lon2 = q.y;

    const auto dLat = (lat2 - lat1) * M_PI / 180.0f;
    const auto dLon = (lon2 - lon1) * M_PI / 180.0f;

    // convert to radians
    lat1 = lat1 * M_PI / 180.0f;
    lat2 = lat2 * M_PI / 180.0f;

    // apply formula
    float a = powf(sinf(dLat / 2), 2) +
              powf(sinf(dLon / 2), 2) * cosf(lat1) * cosf(lat2);
    constexpr float rad = 6371;
    float c = 2 * asinf(sqrtf(a));
    return rad * c;
  }
};
}  // namespace Distance

template <typename DataT, typename ResultT, int block_threads,
          typename ReductionOp>
__device__ void FunctionKernel(cg::thread_group g, DataT* u_buffer,
                               ResultT* u_result, const DataT q) {
  // This need to be a conexpr, because I am passing this as a template argument
  // for Cub library. Although is is just the same as 'g.size()'
  // constexpr int block_threads = 1024;
  constexpr int items_to_reduce = 1024;
  constexpr int items_per_thread = items_to_reduce / block_threads;

  using BlockLoad = hipcub::BlockLoad<DataT, block_threads, items_per_thread,
                                   hipcub::BLOCK_LOAD_STRIPED>;
  using BlockReduce = hipcub::BlockReduce<ResultT, block_threads>;

  constexpr auto functor = ReductionOp();

  __shared__ union {
    typename BlockLoad::TempStorage load;
    typename BlockReduce::TempStorage reduce;
  } temp_storage;

  const auto tid = g.thread_rank();

  DataT thread_data[items_per_thread];
  ResultT thread_value[items_per_thread];

  BlockLoad(temp_storage.load).Load(u_buffer, thread_data);

#pragma unroll
  for (int i = 0; i < items_per_thread; ++i) {
    thread_value[i] = functor(thread_data[i], q);
  }

  ResultT aggregate =
      BlockReduce(temp_storage.reduce).Reduce(thread_value, hipcub::Min());

  // Final step reduction
  if (tid == 0) u_result[0] = min(u_result[0], aggregate);
}

__device__ __forceinline__ void WaitCPU(volatile int* com) {
  int block_id = blockIdx.x;
  while (com[block_id] != 1 && com[kNumBlocks] != 1) {
    __threadfence_system();
  }
}

__device__ __forceinline__ void WorkComplete(volatile int* com) {
  int block_id = blockIdx.x;
  com[block_id] = 0;
}

template <typename DataT, typename ResultT, int block_threads = 1024,
          typename ReductionOp = Distance::EuclideanFunctor>
__global__ void PersistentKernel(DataT* u_buffer, const DataT q,
                                 ResultT* u_result, volatile int* com) {
  auto cta = cg::this_thread_block();
  const auto tid = cta.thread_rank();

  while (com[kNumBlocks] != 1) {
    if (tid == 0) WaitCPU(com);
    __syncthreads();

    // cancelling point
    if (com[kNumBlocks] == 1) return;

    FunctionKernel<DataT, ResultT, block_threads, ReductionOp>(cta, u_buffer,
                                                               u_result, q);

    if (tid == 0) WorkComplete(com);
  }
}

template <typename DataT, typename ResultT, int block_threads = 1024,
          typename ReductionOp = Distance::EuclideanFunctor>
__global__ void NormalKernel(DataT* d_data, const int n, const DataT q,
                             ResultT* u_result) {
  auto cta = cg::this_thread_block();
  const auto tid = cta.thread_rank();
  // const int block_threads = cta.size();

  const auto iterations = n / block_threads;
  for (int i = 0; i < iterations; ++i) {
    // if (tid == 0) printf("gpu iteration (%d/%d)\n", i, iterations);
    FunctionKernel<DataT, ResultT, block_threads, ReductionOp>(
        cta, d_data + i * block_threads, u_result, q);
  }
}

void StartGPU(int* com) {
  // atomic?
  for (int i = 0; i < kNumBlocks; ++i) com[i] = 1;
}

void WaitGPU(int* com) {
  int sum;
  do {
    sum = 0;
    asm volatile("" ::: "memory");
    for (int i = 0; i < kNumBlocks; ++i) sum |= com[i];
  } while (sum != 0);
}

void EndGPU(int* com) {
  printf("cpu is ending GPU\n");
  com[kNumBlocks] = 1;
}

static float2 RandomPoint() {
  static thread_local std::mt19937 generator(114514);  // NOLINT(cert-msc51-cpp)
  static std::uniform_real_distribution<float> lat_dist(-90.0f, 90.0f);
  static std::uniform_real_distribution<float> lon_dist(-180.0f, 180.0f);

  return {lat_dist(generator), lon_dist(generator)};
}

std::vector<float2> GenerateRandomPoints(const int num_points) {
  std::vector<float2> points(num_points);
  std::generate(points.begin(), points.end(), RandomPoint);
  return points;
}

float2* tmp = nullptr;

int main(int argc, char** argv) {
  if (argc < 2) {
    std::cerr << "Bad argument\n";
    return EXIT_FAILURE;
  }

  const int enable_pk = atoi(argv[1]);

  constexpr int n = 1024 * 1024;
  constexpr int buffer_size = 1024;
  constexpr auto num_threads = 1024;

  float2* u_buffer = nullptr;
  float* u_result = nullptr;
  int* u_com = nullptr;

  cudaAllocMapped(&u_buffer, sizeof(float2) * buffer_size);
  cudaAllocMapped(&u_result, sizeof(float) * 1);
  cudaAllocMapped(&u_com, sizeof(int) * (kNumBlocks + 1));

  u_result[0] = std::numeric_limits<float>::max();

  auto h_p_data = GenerateRandomPoints(n);
  const float2 q{0.0f, 0.0f};

  TimeTask("CPU Compute: ", [&] {
    auto sum = std::numeric_limits<float>::max();

    constexpr auto functor = Distance::EuclideanFunctor();
    for (int i = 0; i < n; ++i) {
      const auto dist = functor(h_p_data[i], q);
      sum = std::min(sum, dist);
    }

    std::cout << "Ground truth: " << sum << std::endl;
  });

  tmp = (float2*)malloc(sizeof(float2) * n);
  TimeTask("CPU Memcpy: ",
           [&] { memcpy(tmp, h_p_data.data(), sizeof(float2) * n); });

  if (enable_pk) {
    // Launching the PK only once
    PersistentKernel<<<kNumBlocks, num_threads>>>(u_buffer, q, u_result, u_com);

    TimeTask("PK GPU: ", [&] {
      const auto iterations = n / num_threads;
      for (int i = 0; i < iterations; ++i) {
        // std::cout << "\nIteration: (" << i << '/' << iterations << ')' <<
        // std::endl;
        memcpy(u_buffer, h_p_data.data() + i * num_threads,
               sizeof(float2) * num_threads);

        StartGPU(u_com);

        // CPU Works here...

        WaitGPU(u_com);
      }
    });

    EndGPU(u_com);

    HANDLE_ERROR(hipHostFree(u_buffer));
    HANDLE_ERROR(hipHostFree(u_com));

  } else {
    float2* d_data = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&d_data, sizeof(float2) * n));
    HANDLE_ERROR(hipDeviceSynchronize());

    TimeTask("Normal GPU (1 block) memcpy: ", [&] {
      HANDLE_ERROR(hipMemcpy(d_data, h_p_data.data(), sizeof(float2) * n,
                              hipMemcpyHostToDevice));
    });

    TimeTask("Normal GPU (1 block) compute: ", [&] {
      NormalKernel<float2, float>
          <<<kNumBlocks, num_threads>>>(d_data, n, q, u_result);
      HANDLE_ERROR(hipGetLastError());

      HANDLE_ERROR(hipDeviceSynchronize());
    });
  }

  std::cout << "\tu_result: " << u_result[0] << std::endl;

  HANDLE_ERROR(hipHostFree(u_result));

  return 0;
}