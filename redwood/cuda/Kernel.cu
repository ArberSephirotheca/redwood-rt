#include "hip/hip_runtime.h"
#include <>

#include <limits>

#include "../Kernel.hpp"
#include "CudaUtils.cuh"
#include "hip/hip_runtime.h"

hipStream_t streams[kNumStreams];
bool stream_created = false;

// Global variable
// Need to be registered
const Point4F* usm_leaf_node_table = nullptr;

__global__ void CudaWarmup() {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

namespace redwood::internal {

void BackendInitialization() {
  CudaWarmup<<<1, 1024>>>();
  HANDLE_ERROR(hipDeviceSynchronize());
}

void RegisterLeafNodeTable(const void* leaf_node_table,
                           const int num_leaf_nodes) {
  usm_leaf_node_table = static_cast<const Point4F*>(leaf_node_table);
}

// CUDA Only
void AttachStreamMem(const int stream_id, void* addr) {
  if (!stream_created) {
    for (unsigned i = 0; i < kNumStreams; i++) {
      HANDLE_ERROR(hipStreamCreate(&streams[i]));
    }
    stream_created = true;
  }

  hipStreamAttachMemAsync(streams[stream_id], addr);
}

void DeviceSynchronize() { HANDLE_ERROR(hipDeviceSynchronize()); }

void DeviceStreamSynchronize(const int stream_id) {
  HANDLE_ERROR(hipStreamSynchronize(streams[stream_id]));
}

}  // namespace redwood::internal