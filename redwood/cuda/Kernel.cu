#include "hip/hip_runtime.h"
#include <>

#include <limits>

#include "../Kernel.hpp"
#include "CudaUtils.cuh"
#include "hip/hip_runtime.h"

hipStream_t streams[kNumStreams];
bool stream_created = false;

// Global variable
// Need to be registered
// const Point4F* usm_leaf_node_table = nullptr;

__global__ void CudaWarmup() {
  const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

namespace redwood::internal {

void BackendInitialization() {
  CudaWarmup<<<1, 1024>>>();
  HANDLE_ERROR(hipDeviceSynchronize());
}

// CUDA Only
void AttachStreamMem(const int stream_id, void* addr) {
  if (!stream_created) {
    for (unsigned i = 0; i < kNumStreams; i++) {
      HANDLE_ERROR(hipStreamCreate(&streams[i]));
    }
    stream_created = true;
  }

  hipStreamAttachMemAsync(streams[stream_id], addr);
}

void DeviceSynchronize() { HANDLE_ERROR(hipDeviceSynchronize()); }

void DeviceStreamSynchronize(const int stream_id) {
  HANDLE_ERROR(hipStreamSynchronize(streams[stream_id]));
}

}  // namespace redwood::internal